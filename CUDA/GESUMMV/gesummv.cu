#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gesummv.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

//#define RUN_ON_CPU


void gesummv(DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n), DATA_TYPE POLYBENCH_1D(x,N,n), 
		DATA_TYPE POLYBENCH_1D(y,N,n), DATA_TYPE POLYBENCH_1D(tmp,N,n))
{
	int i, j;
	
	for (i = 0; i < N; i++)
	{
		tmp[i] = 0;
		y[i] = 0;
		for (j = 0; j < N; j++)
		{
			tmp[i] = A[i][j] * x[j] + tmp[i];
			y[i] = B[i][j] * x[j] + y[i];
		}
		
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}


void init(DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_1D(x,N,n))
{
  	int i, j;

 	for (i = 0; i < N; i++)
    	{
    		x[i] = ((DATA_TYPE) i) / N;
      	
		for (j = 0; j < N; j++) 
		{
			A[i][j] = ((DATA_TYPE) i*j) / N;
		}
    }
}


void compareResults(DATA_TYPE POLYBENCH_1D(y,N,n), DATA_TYPE POLYBENCH_1D(y_outputFromGpu,N,n))
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<(N); i++) 
	{
		if (percentDiff(y[i], y_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gesummv_kernel(DATA_TYPE* a, DATA_TYPE* b, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j = 0; j < N; j++)
		{	
			tmp[i] += a[i * N + j] * x[j];
			y[i] += b[i * N + j] * x[j];
		}
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}

void gesummvCuda(DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n), DATA_TYPE POLYBENCH_1D(x,N,n), DATA_TYPE POLYBENCH_1D(y,N,n), DATA_TYPE POLYBENCH_1D(tmp,N,n), DATA_TYPE POLYBENCH_1D(y_outputFromGpu,N,n))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * N);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), 1);


	/* Start timer. */
  	polybench_start_instruments;

	gesummv_kernel<<< grid, block>>>(A_gpu,B_gpu,x_gpu, y_gpu, tmp_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(y,N,n))

{
  int i;

  for (i = 0; i < n; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, y[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
}


int main(int argc, char *argv[])
{
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,N,N,n,n);
	POLYBENCH_1D_ARRAY_DECL(x,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(y,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(y_outputFromGpu,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(tmp,DATA_TYPE,N,n);

	init(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(x));
	
	GPU_argv_init();
	gesummvCuda(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(tmp), 
		POLYBENCH_ARRAY(y_outputFromGpu));
	
	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		gesummv(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(tmp));
		
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(y_outputFromGpu));

	#else //print output to stderr so no dead code elimination

		print_array(N, POLYBENCH_ARRAY(y_outputFromGpu));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);  
	POLYBENCH_FREE_ARRAY(x);  
	POLYBENCH_FREE_ARRAY(y);
	POLYBENCH_FREE_ARRAY(y_outputFromGpu);
	POLYBENCH_FREE_ARRAY(tmp);

	return 0;
}

#include "../../common/polybench.c"
