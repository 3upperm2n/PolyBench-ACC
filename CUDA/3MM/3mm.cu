#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "3mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//#define RUN_ON_CPU


void init_array(DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(D, NI, NJ, ni, nj))
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}
  
	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i][j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}
  
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NM; j++)
		{
			C[i][j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}
  
	for (i = 0; i < NM; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i][j] = ((DATA_TYPE) i*(j+2)) / NK;
		}
	}
}


void compareResults(DATA_TYPE POLYBENCH_2D(G, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(G_outputFromGpu, NI, NJ, ni, nj))
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NI; i++)
	{
		for (j=0; j < NL; j++)
		{
			if (percentDiff(G[i][j], G_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;				
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

	
__global__ void mm3_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{
		int k;
		for(k=0; k < NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}

	
__global__ void mm3_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NJ) && (j < NL))
	{
		int k;
		for(k=0; k < NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{
		int k;
		for(k=0; k < NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
		}
	}
}


void mm3_cpu(DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(D, NI, NJ, ni, nj), 
	DATA_TYPE POLYBENCH_2D(E, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(F, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(G, NI, NJ, ni, nj))
{
	int i,j,k;
	
	/* E := A*B */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			E[i][j] = 0;
			for (k = 0; k < NK; ++k)
			{
				E[i][j] += A[i][k] * B[k][j];
			}
		}
	}
		
	/* F := C*D */
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NL; j++)
		{
			F[i][j] = 0;
			for (k = 0; k < NM; ++k)
			{
				F[i][j] += C[i][k] * D[k][j];
			}
		}
	}

  	/* G := E*F */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			G[i][j] = 0;
			for (k = 0; k < NJ; ++k)
			{
				G[i][j] += E[i][k] * F[k][j];
			}
		}
	}
}


void mm3Cuda(DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(D, NI, NJ, ni, nj), 
	DATA_TYPE POLYBENCH_2D(E, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(F, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(G, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(G_outputFromGpu, NI, NJ, ni, nj))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;
	
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NJ * NM);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NM * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&F_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&G_gpu, sizeof(DATA_TYPE) * NI * NL);

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, F, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(G_gpu, G, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));

	/* Start timer. */
  	polybench_start_instruments;

	mm3_kernel1<<<grid1,block>>>(A_gpu, B_gpu, E_gpu);
	hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block>>>(C_gpu, D_gpu, F_gpu);
	hipDeviceSynchronize();
	mm3_kernel3<<<grid3,block>>>(E_gpu, F_gpu, G_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;
	hipMemcpy(G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nl,
		 DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, G[i][j]);
	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


int main(int argc, char** argv)
{
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(D,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(E,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(F,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(G,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(G_outputFromGpu,DATA_TYPE,NI,NJ,ni,nj);

	init_array(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));

	GPU_argv_init();

	mm3Cuda(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		mm3_cpu(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G));
	
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

	#else //print output to stderr so no dead code elimination

		print_array(NI, NJ, POLYBENCH_ARRAY(G_outputFromGpu));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(D);
	POLYBENCH_FREE_ARRAY(E);
	POLYBENCH_FREE_ARRAY(F);
	POLYBENCH_FREE_ARRAY(G);
	POLYBENCH_FREE_ARRAY(G_outputFromGpu);

	return 0;
}

#include "../../common/polybench.c"

