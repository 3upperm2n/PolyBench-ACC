#include "hip/hip_runtime.h"
/**
 * lu.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>

#define POLYBENCH_TIME 1

#include "lu.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

//#define RUN_ON_CPU


void lu(DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
	for (int k = 0; k < N; k++)
    {
		for (int j = k + 1; j < N; j++)
		{
			A[k][j] = A[k][j] / A[k][k];
		}

		for (int i = k + 1; i < N; i++)
		{
			for (int j = k + 1; j < N; j++)
			{
				A[i][j] = A[i][j] - A[i][k] * A[k][j];
			}
		}
    }
}


void init_array(DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
	int i, j;

	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j + 1) / N;
		}
	}
}


void compareResults(DATA_TYPE POLYBENCH_2D(A_cpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n))
{
	int i, j, fail;
	fail = 0;
	
	// Compare a and b
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
			if (percentDiff(A_cpu[i][j], A_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void lu_kernel1(DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((j > k) && (j < N))
	{
		A[k*N + j] = A[k*N + j] / A[k*N + k];
	}
}


__global__ void lu_kernel2(DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i > k) && (j > k) && (i < N) && (j < N))
	{
		A[i*N + j] = A[i*N + j] - A[i*N + k] * A[k*N + j];
	}
}


void luCuda(DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n))
{
	DATA_TYPE* AGpu;

	hipMalloc(&AGpu, N * N * sizeof(DATA_TYPE));
	hipMemcpy(AGpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid1(1, 1, 1);
	dim3 grid2(1, 1, 1);

	/* Start timer. */
  	polybench_start_instruments;

	for (int k = 0; k < N; k++)
	{
		grid1.x = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block1.x)));
		lu_kernel1<<<grid1, block1>>>(AGpu, k);
		hipDeviceSynchronize();

		grid2.x = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block2.x)));
		grid2.y = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block2.y)));
		lu_kernel2<<<grid2, block2>>>(AGpu, k);
		hipDeviceSynchronize();
	}
	
	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(A_outputFromGpu, AGpu, N * N * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
	hipFree(AGpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      fprintf (stderr, DATA_PRINTF_MODIFIER, A[i][j]);
      if ((i * n + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}
	

int main(int argc, char *argv[])
{
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,N,n,n);
  	POLYBENCH_2D_ARRAY_DECL(A_outputFromGpu,DATA_TYPE,N,N,n,n);

	init_array(POLYBENCH_ARRAY(A));

	GPU_argv_init();
	luCuda(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_outputFromGpu));
	

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		lu(POLYBENCH_ARRAY(A));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_outputFromGpu));

	#else //print output to stderr so no dead code elimination

		print_array(N, POLYBENCH_ARRAY(A_outputFromGpu));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(A_outputFromGpu);

   	return 0;
}

#include "../../common/polybench.c"

