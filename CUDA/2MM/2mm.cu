#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "2mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

//#define RUN_ON_CPU


void init_array(DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk), DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj), 
		DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}

	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i][j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}

	for (i = 0; i < NL; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i][j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i][j] = ((DATA_TYPE) i*(j+2)) / NK;	
		}
	}
}


void compareResults(DATA_TYPE POLYBENCH_2D(E, NL, NI, nl, ni), DATA_TYPE POLYBENCH_2D(E_outputFromGpu, NL, NI, nl, ni))
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NL; i++)
	{
		for (j=0; j < NI; j++)
		{
			if (percentDiff(E[i][j], E_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void mm2_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{ 
		int k;
		for (k = 0; k < NK; k++)
		{
			C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}


__global__ void mm2_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{ 
		int k;
		for (k = 0; k < NJ; k++)
		{
			E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
		}
	}
}


void mm2_cpu(DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nl), DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj), DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl), 
		DATA_TYPE POLYBENCH_2D(E,NI,NL,ni,nl))
{
	int i, j, k;
	
  	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i][j] = 0.0;
			for (k = 0; k < NK; ++k)
			{
				C[i][j] += A[i][k] * B[k][j];
			}
		}
	}
	
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			E[i][j] = 0.0;
			for (k = 0; k < NJ; ++k)
			{
				E[i][j] += C[i][k] * D[k][j];
			}
		}
	}
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nl,
		 DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, D[i][j]);
	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


void mm2Cuda(DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nl), DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj), DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl), 
		DATA_TYPE POLYBENCH_2D(E,NI,NL,ni,nl), DATA_TYPE POLYBENCH_2D(E_outputFromGpu,NI,NL,ni,nl))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NL);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
		
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)ceil( ((float)NJ) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	dim3 grid2((size_t)ceil( ((float)NL) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );

	/* Start timer. */
  	polybench_start_instruments;

	mm2_kernel1<<<grid1,block>>>(A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();
	mm2_kernel2<<<grid2,block>>>(C_gpu, D_gpu, E_gpu);
	hipDeviceSynchronize();

	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
}


int main(int argc, char** argv)
{
	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);
	POLYBENCH_2D_ARRAY_DECL(D,DATA_TYPE,NJ,NL,nj,nl);
	POLYBENCH_2D_ARRAY_DECL(E,DATA_TYPE,NI,NL,ni,nl);
	POLYBENCH_2D_ARRAY_DECL(E_outputFromGpu,DATA_TYPE,NI,NL,ni,nl);
	
  	init_array(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));
	GPU_argv_init();

	mm2Cuda(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), POLYBENCH_ARRAY(E_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		mm2_cpu(POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E));

		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(POLYBENCH_ARRAY(E), POLYBENCH_ARRAY(E_outputFromGpu));

	#else //print output to stderr so no dead code elimination

		print_array(NI, NL, POLYBENCH_ARRAY(E_outputFromGpu));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(D);
	POLYBENCH_FREE_ARRAY(E);
	POLYBENCH_FREE_ARRAY(E_outputFromGpu);

  	return 0;
}

#include "../../common/polybench.c"
